#include "hip/hip_runtime.h"
#include "gpu-mc.h"
#include "gpu-mc-kernel.h"
#include <utility>
#include <vector>
#include <iostream>
#include <cassert>
#include <cuda_gl_interop.h>
#include <cmath>
#include <ctime>

const unsigned int CUBESIZE = 8;
const unsigned int LOG2CUBESIZE = 3;
const unsigned int CUBESIZEHP = 2;
const unsigned int LOG2CUBESIZEHP = 1;
unsigned int SIZE;
unsigned int rawMemSize;
unsigned char * rawDataPtr;
struct hipGraphicsResource * vbo_cuda = NULL;
GLuint vbo_gl = 0;
size_t vbo_size = 0;

unsigned int sum_of_triangles = 0;

// How to use the VBO:
//      1. calc number of triangles
//      2. resize VBO to the correct size (triangles and normals)
//      3. calc the triangles
//      4. render

// first level has char4 as datatype, which contains: (number of triangles, cube index, value of first cube element, 0)
// first to second level contain volumes with unsigned char as elements
// third to fifth (including) level contain unsigned short as elements
// sixth level and more uses int
std::vector<std::pair<hipExtent, hipPitchedPtr> > images_size_pointer;

int isolevel = 49;

template<typename T>
T log2(T val) {
    T log2Val = 0;
    while (val > 1) {
      val /= 2; log2Val++;
    }
    return log2Val;
}

bool handleCudaError(const hipError_t& status) {
    std::string error_msg;
    switch (status) {
        case hipErrorInvalidValue: {
            error_msg = "hipErrorInvalidValue";
            break;
        }
        case hipErrorInvalidDevicePointer: {
            error_msg = "cudaErrorInvalidDevidePointer";
            break;
        }
        case hipErrorInvalidSymbol: {
            error_msg = "hipErrorInvalidSymbol";
            break;
        }
        case hipErrorInvalidMemcpyDirection: {
            error_msg = "hipErrorInvalidMemcpyDirection";
            break;
        }
        case hipErrorInvalidDevice: {
            error_msg = "hipErrorInvalidDevice";
            break;
        }
        case hipErrorSetOnActiveProcess: {
            error_msg = "hipErrorSetOnActiveProcess";
            break;
        }
        case hipErrorInvalidHandle: {
            error_msg = "hipErrorInvalidHandle";
            break;
        }
        case hipErrorOutOfMemory: {
            error_msg = hipErrorOutOfMemory;
            break;
        }
        case hipErrorUnknown: {
            error_msg = "hipErrorUnknown";
            break;
        }
        case hipErrorLaunchTimeOut: {
            error_msg = "hipErrorLaunchTimeOut";
            break;
        }
        case hipErrorNoDevice: {
            error_msg = "hipErrorNoDevice";
            break;
        }
        case hipSuccess: {
            break;
        }
        case hipErrorLaunchFailure: {
            error_msg = "hipErrorLaunchFailure";
            break;
        }
        case hipErrorInvalidDeviceFunction: {
            error_msg = "hipErrorInvalidDeviceFunction";
            break;
        }
        case hipErrorLaunchOutOfResources: {
            error_msg= "hipErrorLaunchOutOfResources";
            break;
        }
        default: {
            error_msg = "unknown error";
            break;
        }
    }
    
    if (status != hipSuccess) {
        std::cout << "!!!!!!!!! " << error_msg << ", " << hipGetErrorString(status) << " !!!!!!!!!" << std::endl;
        exit(1);
    }

    return status != hipSuccess;
}

void setupCuda(unsigned char * voxels, unsigned int size, GLuint vbo) {
    vbo_gl = vbo;
    if (vbo != 0)
        handleCudaError(cudaGLSetGLDevice(0));

    SIZE = size;

    // Create images for the HistogramPyramid
    hipExtent bufferSize;
    hipPitchedPtr tmpDataPtr;
    // Make the two first buffers use INT8
    bufferSize.width = SIZE * sizeof(uchar4);
    bufferSize.height = SIZE;
    bufferSize.depth = SIZE;
    handleCudaError(hipMalloc3D(&tmpDataPtr, bufferSize));
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    bufferSize.width = bufferSize.depth/2 * sizeof(uchar1);
    bufferSize.height = bufferSize.depth/2;
    bufferSize.depth = bufferSize.depth/2;
    handleCudaError(hipMalloc3D(&tmpDataPtr, bufferSize));
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    // And the third, fourth and fifth INT16
    for (unsigned int i = 0; i < 3; i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(ushort1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        handleCudaError(hipMalloc3D(&tmpDataPtr, bufferSize));
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // The rest will use INT32
    for(unsigned int i = 5; i < (log2(SIZE)); i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(uint1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        // Image cant be 1x1x1
        if (bufferSize.depth == 1) {
            bufferSize.width = 2 * sizeof(uint1);
            bufferSize.height = 2;
            bufferSize.depth = 2;
        }
        handleCudaError(hipMalloc3D(&tmpDataPtr, bufferSize));
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // copy cudapitchedPtr to device
    for (unsigned int i = 0; i < images_size_pointer.size(); i++) {
        handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("levels"), &(images_size_pointer.at(i).second), sizeof(hipPitchedPtr), i*sizeof(hipPitchedPtr), hipMemcpyHostToDevice));
    }

    // Transfer dataset to device
    rawMemSize = SIZE*SIZE*SIZE*sizeof(unsigned char);
    handleCudaError(hipMalloc((void **) &rawDataPtr, rawMemSize));
    handleCudaError(hipMemcpy(rawDataPtr, voxels, rawMemSize, hipMemcpyHostToDevice));
//    delete[] voxels;
}

// classifies each voxel and calculates the number of triangles needed for this
// voxel
void updateScalarField() {
    hipExtent _size = images_size_pointer.at(0).first;
    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((_size.depth / CUBESIZE) * (_size.depth / CUBESIZE), _size.depth / CUBESIZE, 1);
    int log2GridSize = log2(_size.depth / CUBESIZE);
    kernelClassifyCubes<<<grid , block>>>(images_size_pointer.at(0).second, rawDataPtr, isolevel, log2GridSize, _size.depth/CUBESIZE-1, LOG2CUBESIZE, _size.depth);
    handleCudaError(hipGetLastError());
    hipDeviceSynchronize();
}

#ifdef DEBUG
// code to test classify cubes
unsigned int get_index(unsigned int x, unsigned int y, unsigned int z) {
  return x + y*SIZE + z*SIZE*SIZE;
}

void get_voxel_from_cube_id(unsigned int cube_id, unsigned int *x, unsigned int *y, unsigned *z) {
  // return lower left position of cube, other points can be obtained with +0,1
  *z = cube_id / (SIZE-1) / (SIZE-1);
  unsigned int cube_id_plane = cube_id % ((SIZE-1) * (SIZE-1));
  *y = cube_id_plane / (SIZE-1);
  *x = cube_id_plane % (SIZE-1);
}

int bit2Offset[] = {0, 1, 3, 2, 4, 5, 7, 6};
uint4 lokalCubeOffsets[8] = {
		{0, 0, 0, 0},
		{1, 0, 0, 0},
		{0, 0, 1, 0},
		{1, 0, 1, 0},
		{0, 1, 0, 0},
		{1, 1, 0, 0},
		{0, 1, 1, 0},
		{1, 1, 1, 0},
	}; 

unsigned char lokalNrOfTriangles[256] = {0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2, 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2, 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1, 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1, 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0};

template<typename T>
T* get_data_from_pitched_ptr(hipExtent size, hipPitchedPtr source) {
    T * lvl0_data = new T[size.depth*size.depth*size.depth];
    hipPitchedPtr h_pitched_ptr = make_hipPitchedPtr(lvl0_data, size.depth*sizeof(T), size.depth, size.depth);
    struct hipMemcpy3DParms parms = {0};
    parms.srcPtr = source;
    parms.dstPtr = h_pitched_ptr;
    parms.extent = size;
    parms.kind = hipMemcpyDeviceToHost;
    clock_t start = clock();
    handleCudaError(hipMemcpy3D(&parms));
    std::cout << "get_data_from_pitched_ptr: hipMemcpy took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    return lvl0_data;
}

template<typename T>
T* get_data_from_pitched_ptr(std::pair<hipExtent, hipPitchedPtr> source) {
    return get_data_from_pitched_ptr<T>(source.first, source.second);
}

template<typename T>
T* get_data_from_pitched_ptr(unsigned int level) {
    return get_data_from_pitched_ptr<T>(images_size_pointer.at(level));
}

bool testUpdateScalarField(unsigned char * voxels) {
    updateScalarField();
    // get level0 data from gpu
    uchar4 * lvl0_data = get_data_from_pitched_ptr<uchar4>(0);

    sum_of_triangles = 0;

    // calc for each voxel index and number of triangles using a different implementation
    for (unsigned int i = 0; i < (SIZE-1)*(SIZE-1)*(SIZE-1); i++) {
        // get base voxel of the cube
        unsigned int x, y, z;
        get_voxel_from_cube_id(i, &x, &y, &z);  
        // look which vertices are below or above our threshold
        int lookuptable_index = 0;

        for (unsigned int id = 0; id < 8; id++) {
            uint4 offset = lokalCubeOffsets[bit2Offset[id]];
            unsigned char voxel = voxels[get_index(x + offset.x, y + offset.y, z + offset.z)];
            bool greater = voxel > isolevel;
            lookuptable_index |= greater << id;
        }
        unsigned int num_triangles = lokalNrOfTriangles[lookuptable_index];
        sum_of_triangles += num_triangles;

        // compare with results from gpu
        if (voxels[get_index(x, y, z)] != lvl0_data[get_index(x, y, z)].z) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "vertex value got from raw data is: " << static_cast<int>(voxels[get_index(x, y, z)]) << "\n value saved on gpu is: " << static_cast<int>(lvl0_data[get_index(x, y, z)].z) << std::endl;
            return false;
        }

        if (lookuptable_index != lvl0_data[get_index(x, y, z)].y) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "cube index calculated in software: " << lookuptable_index << "\ncube index calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].y)) << std::endl;
            return false;
        }

        if (num_triangles != lvl0_data[get_index(x, y, z)].x) {
            std::cout << "No match in number of triangles at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "number triangles calculated in software: " << num_triangles << "\nnumber of triangles calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].x)) << std::endl;
            return false;
        }
    }

    std::cout << "you will get " << sum_of_triangles << " triangles" << std::endl;

    delete [] lvl0_data;
    return true;
}
// end of code to test classifycubes
#endif // DEBUG

// calculates the total number of triangles needed
void histoPyramidConstruction() {
    dim3 block(CUBESIZEHP, CUBESIZEHP, CUBESIZEHP);
    
    // i=    0       1        2        3        4      5
    // uchar4, uchar1, ushort1, ushort1, ushort1, uint1, ...
    for (unsigned int i = 0; i < log2(SIZE)-1; i++) {
        hipExtent _size = images_size_pointer.at(i+1).first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        if (i == 0)
            // second level
            // uchar4 -> uchar1
            kernelConstructHPLevel<uchar4, uchar1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP);
        else if (i == 1)
            // third level
            // uchar1 -> ushort1
            kernelConstructHPLevel<uchar1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i == 2 || i == 3)
            // fourth, fifth level
            // ushort1 -> ushort1
            // ushort1 -> ushort1
            kernelConstructHPLevel<ushort1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i == 4)
            // sixth level
            // ushort1 -> uint1
            kernelConstructHPLevel<ushort1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else
            // all other levels
            // uint1 -> uint1
            kernelConstructHPLevel<uint1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        handleCudaError(hipGetLastError());
        hipDeviceSynchronize();
    }
}

template<typename T>
unsigned int sum_3d_array(T const * const _3darray, const hipExtent& dim) {
    unsigned int sum = 0;
    for (unsigned int id = 0; id < dim.depth*dim.depth*dim.depth; id++) {
        sum+= _3darray[id].x;
    }
    return sum;
}

template<typename T>
unsigned int sum_3d_array(const std::pair<hipExtent, hipPitchedPtr>& pair) {
    clock_t start = clock();
    T* sum_of_triangles_from_gpu = get_data_from_pitched_ptr<T>(pair);
    std::cout << "sum_3d_array: get_data_from_pitched_ptr took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    start = clock();
    unsigned int sum = sum_3d_array(sum_of_triangles_from_gpu, pair.first);
    std::cout << "sum_3d_array: sum_3d_array took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    delete [] sum_of_triangles_from_gpu;
    return sum;
}

template<typename T>
unsigned int sum_3d_array(unsigned int level) {
    std::pair<hipExtent, hipPitchedPtr> pair = images_size_pointer.at(level);
    return sum_3d_array<T>(pair);
}

#ifdef DEBUG
template<typename T>
bool templatedTestHistoPyramidConstruction(unsigned int level) {
    unsigned int sum = sum_3d_array<T>(level);
    if (sum != sum_of_triangles) {
        std::cout << "at level " << level << std::endl;
        std::cout << "number of triangles calculated in software and hardware mismatches!" << std::endl;
        std::cout << "software: " << sum_of_triangles << ", hardware: " << sum << std::endl;
    }
    return sum == sum_of_triangles;
}

bool testHistoPyramidConstruction() {
    histoPyramidConstruction();
    bool success = true;
    for (unsigned int i = 0; i < log2(SIZE); i++) {
        if (i == 0)
            success &= templatedTestHistoPyramidConstruction<uchar4>(i);
        else if (i == 1)
            success &= templatedTestHistoPyramidConstruction<uchar1>(i);
        else if (i > 1 && i < 5)
            success &= templatedTestHistoPyramidConstruction<ushort1>(i);
        else
            success &= templatedTestHistoPyramidConstruction<uint1>(i);
    }
    return success;
}
#endif // DEBUG

void resizeVBO(size_t _vbo_size, bool clear) {
    if (vbo_cuda != NULL) {
        handleCudaError(hipGraphicsUnregisterResource(vbo_cuda));
        vbo_cuda = NULL;
    }

    float3 * data = NULL;
    if (clear) {
        data = new float3[_vbo_size/sizeof(float3)];
        for (unsigned int i = 0; i < _vbo_size/sizeof(float3); i++) {
            float3 val = {0};
/*
            if (i % 6 == 0)
                val = make_float3(0.0f, 200.0f, 0.0f);
            if (i % 6 == 1)
                val = make_float3(100.0f, 200.0f, 0.0f);
            if (i % 6 == 2)
                val = make_float3(100.0f, 300.0f, 0.0f);
            if (i % 6 > 2)
                val = make_float3(0.0f, 0.0f, 1.0f);
*/
            data[i] = val;
        }
    }

    glBindBuffer(GL_ARRAY_BUFFER, vbo_gl);
    glBufferData(GL_ARRAY_BUFFER, _vbo_size, data, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    handleCudaError(hipGraphicsGLRegisterBuffer(&vbo_cuda, vbo_gl, cudaGraphicsMapFlagsWriteDiscard));

    delete [] data;
    
    vbo_size = _vbo_size;
}

size_t resizeVBOIfNeeded(bool clear = false) {
    // resize buffer
    // normals, triangles, three coordinates, three points in float
    size_t buffer_size = sum_of_triangles*2*3*3*sizeof(float);
    if (buffer_size > vbo_size && vbo_gl != 0)
        resizeVBO(buffer_size, clear);
    return buffer_size;
}

unsigned int getNumberOfTriangles() {
    unsigned int sum = 0;
    assert(log2(SIZE) == images_size_pointer.size());
    size_t num_of_levels = images_size_pointer.size();
    std::pair<hipExtent, hipPitchedPtr> pair =  images_size_pointer.back();
    clock_t start = clock();
    if (num_of_levels == 1)
        sum = sum_3d_array<uchar4>(pair);
    else if (num_of_levels == 2)
        sum = sum_3d_array<uchar1>(pair);
    else if (num_of_levels == 3 || num_of_levels == 4 || num_of_levels == 5)
        sum = sum_3d_array<ushort1>(pair);
    else
        sum = sum_3d_array<uint1>(pair);
    std::cout << "getNumberOfTriangles: sum_3d_array took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    sum_of_triangles = sum;
    std::cout << "you will get " << sum << " triangles" << std::endl;

    start = clock();
    handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("num_of_levels"), &num_of_levels, sizeof(size_t), 0, hipMemcpyHostToDevice));
    std::cout << "getNumberOfTriangles: copying number of levels to gpu took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    return sum;
}

float3 * getTriangleDataPointer() {
    size_t buffer_size = resizeVBOIfNeeded(true);

    float3 * triangle_data = NULL;
    if (vbo_gl != 0) {
        handleCudaError(hipGraphicsMapResources(1, &vbo_cuda, 0));
        size_t num_bytes = 0;
        handleCudaError(hipGraphicsResourceGetMappedPointer((void**)&triangle_data, &num_bytes, vbo_cuda));
        assert(num_bytes >= buffer_size);
    } else {
        handleCudaError(hipMalloc(&triangle_data, buffer_size));
    }

    return triangle_data;
}

void freeResources(float3 * triangle_data) {
    if (vbo_gl != 0)
        handleCudaError(hipGraphicsUnmapResources(1, &vbo_cuda, 0));
    else
        handleCudaError(hipFree(triangle_data));
}

// creates the VBO
int histoPyramidTraversal() {
    clock_t start = clock();
    getNumberOfTriangles();
    std::cout << "histoPyramidTraversal: getting number of triangles took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";

    start = clock();
    float3 * triangle_data = getTriangleDataPointer();
    assert(triangle_data != NULL);
    std::cout << "histoPyramidTraversal: allocating VBO took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";

    // TODO ask device properties how much threads can be started
    //      there are cards, where not all 512 threads can be started, when
    //      some data is in memory
    unsigned int tmp_cube_size = CUBESIZE/2;
    dim3 block(tmp_cube_size, tmp_cube_size, tmp_cube_size);
    int number_of_blocks = sum_of_triangles/tmp_cube_size/tmp_cube_size/tmp_cube_size;
    int grid_dim_x = floor(sqrt(number_of_blocks));
    int grid_dim_y = ceil(number_of_blocks/grid_dim_x);
    dim3 grid(grid_dim_x, grid_dim_y, 1);
    
    start = clock();
    traverseHP<<<grid, block>>>(
        triangle_data,
        isolevel,
        sum_of_triangles,
        log2(SIZE),
        SIZE,
        log2(tmp_cube_size)
        );
    handleCudaError(hipGetLastError());
    std::cout << "histoPyramidTraversal: traverseHP took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    hipDeviceSynchronize();
    
    freeResources(triangle_data);
    return sum_of_triangles;
}

#ifdef DEBUG
bool testCudaPitchedPtrOnDevice() {
    dim3 grid(1,1,1);
    dim3 block(1,1,1);
    bool h_success = true;
    bool * d_success;
    handleCudaError(hipMalloc(&d_success, sizeof(bool)));
    bool success = true;
    unsigned int i = 0;
    for (std::vector<std::pair<hipExtent, hipPitchedPtr> >::iterator iter = images_size_pointer.begin(); iter != images_size_pointer.end(); iter++, i++) {
        std::pair<hipExtent, hipPitchedPtr> pair = *iter;
        cmp_pitched_ptr<<<grid, block>>>(i, pair.second, d_success);
        handleCudaError(hipGetLastError());
        hipMemcpy(&h_success, d_success, sizeof(bool), hipMemcpyDeviceToHost);
        if (!h_success) {
            std::cout << "something is wrong with the hipPitchedPtr copied to the GPU via hipMemcpyToSymbol at level " << i << std::endl;
        }
        success &= h_success;
    }
    handleCudaError(hipFree(d_success));
    return success;
}

bool testHistoPyramidTraversal() {
    histoPyramidTraversal();
    bool success = true;
    size_t num_of_levels_readback = 0;
    handleCudaError(hipMemcpyFromSymbol(&num_of_levels_readback, HIP_SYMBOL("num_of_levels"), sizeof(size_t), 0, hipMemcpyDeviceToHost));

    success &= images_size_pointer.size() == num_of_levels_readback;
    if (!success) {
        std::cout << "number of levels on GPU are not set correctly: " << num_of_levels_readback << ", should be: " << images_size_pointer.size() << std::endl;
    }

    hipPitchedPtr cpp = {0};
    for (unsigned int i = 0; i < num_of_levels_readback; i++) {
        handleCudaError(hipMemcpyFromSymbol(&cpp, HIP_SYMBOL("levels"), sizeof(hipPitchedPtr), i*sizeof(hipPitchedPtr), hipMemcpyDeviceToHost));
        bool tmp_success = cpp == images_size_pointer.at(i).second;
        if (!tmp_success) {
            std::cout << "hipPitchedPtr used as a argument for a kernel on the GPU does not match at level " << i << std::endl;
        }
        success &= tmp_success;
    }
    success &= testCudaPitchedPtrOnDevice();
    return success;
}


bool runTests(unsigned char * voxels) {
  bool success = testUpdateScalarField(voxels);
  success &= testHistoPyramidConstruction();
  success &= testHistoPyramidTraversal();
//  hipDeviceReset();

  return success;
}
#endif // DEBUG

int marching_cube(int _isolevel) {
    if (isolevel != _isolevel) {
        isolevel = _isolevel; 
        clock_t start = clock();
        // first level
        updateScalarField();
        std::cout << "updateScalarField took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";

        // all other levels
        start = clock();
        histoPyramidConstruction();
        std::cout << "histoPyramidConstruction took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";

        start = clock();
        histoPyramidTraversal();
        std::cout << "histoPyramidTraversal took " << static_cast<double>(clock()-start)/CLOCKS_PER_SEC << " seconds\n";
    }
    return sum_of_triangles;
}
