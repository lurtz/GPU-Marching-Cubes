#include "hip/hip_runtime.h"
#include "gpu-mc.h"
#include "gpu-mc-kernel.h"
#include <utility>
#include <vector>

const unsigned int CUBESIZE = 8;
const unsigned int LOG2CUBESIZE = 3;
const unsigned int CUBESIZEHP = 2;
const unsigned int LOG2CUBESIZEHP = 1;
unsigned int SIZE;
unsigned int rawMemSize;
unsigned char * rawDataPtr;

// first level has char4 as datatype, which contains: (number of triangles, cube index, value of first cube element, 0)
// first to second level contain volumes with unsigned char as elements
// third to fifth (including) level contain unsigned short as elements
// sixth level and more uses int
std::vector<std::pair<hipExtent, hipPitchedPtr> > images_size_pointer;

int isolevel = 50;

void setupCuda(unsigned char * voxels, unsigned int size) {
    SIZE = size;

    // Create images for the HistogramPyramid
    hipExtent bufferSize;
    hipPitchedPtr tmpDataPtr;
    // Make the two first buffers use INT8
    bufferSize.width = SIZE * sizeof(uchar4);
    bufferSize.height = SIZE;
    bufferSize.depth = SIZE;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    bufferSize.width = bufferSize.depth/2 * sizeof(uchar1);
    bufferSize.height = bufferSize.depth/2;
    bufferSize.depth = bufferSize.depth/2;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    // And the third, fourth and fifth INT16
    for (unsigned int i = 0; i < 3; i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(ushort1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        hipMalloc3D(&tmpDataPtr, bufferSize);
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // The rest will use INT32
    for(int i = 5; i < (log2((float)SIZE)); i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(uint1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        // Image cant be 1x1x1
        if (bufferSize.depth == 1) {
            bufferSize.width = 2 * sizeof(uint1);
            bufferSize.height = 2;
            bufferSize.depth = 2;
        }
        hipMalloc3D(&tmpDataPtr, bufferSize);
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // Transfer dataset to device
    rawMemSize = SIZE*SIZE*SIZE*sizeof(unsigned char);
    hipMalloc((void **) &rawDataPtr, rawMemSize);
    hipMemcpy(rawDataPtr, voxels, rawMemSize, hipMemcpyHostToDevice);
    delete[] voxels;
}

template<typename T>
T log2(T val) {
    T log2Val = 0;
    while (val > 1) {
      val /= 2; log2Val++;
    }
    return log2Val;
}

void updateScalarField() {
    hipExtent _size = images_size_pointer[0].first;
    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((_size.depth / CUBESIZE) * (_size.depth / CUBESIZE), _size.depth / CUBESIZE, 1);
    int log2GridSize = log2(_size.depth / CUBESIZE);
    kernelClassifyCubes<<<grid , block>>>(images_size_pointer[0].second, rawDataPtr, isolevel, log2GridSize, _size.depth/CUBESIZE-1, LOG2CUBESIZE, _size.depth);
}

void histoPyramidConstruction() {
    // first level
    updateScalarField();

    dim3 block(CUBESIZEHP, CUBESIZEHP, CUBESIZEHP);
    
    unsigned int i = 0;
    // second level
    if (i < log2((float)SIZE)-1) {
        hipExtent _size = images_size_pointer[i+1].first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        kernelConstructHPLevel<uchar4, uchar1><<<grid, block>>>(images_size_pointer[i].second , images_size_pointer[i+1].second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
    i++;

    // third level
    if (i < log2((float)SIZE)-1) {
        hipExtent _size = images_size_pointer[i+1].first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        kernelConstructHPLevel<uchar1, ushort1><<<grid, block>>>(images_size_pointer[i].second , images_size_pointer[i+1].second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
    i++;

    // fourth, fifth level
    for (unsigned int j = 0; i < log2((float)SIZE)-1 && j < 2; i++, j++) {
        hipExtent _size = images_size_pointer[i+1].first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        kernelConstructHPLevel<ushort1, ushort1><<<grid, block>>>(images_size_pointer[i].second , images_size_pointer[i+1].second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }

    // sixth level
    if (i < log2((float)SIZE)-1) {
        hipExtent _size = images_size_pointer[i+1].first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        kernelConstructHPLevel<ushort4, uint1><<<grid, block>>>(images_size_pointer[i].second , images_size_pointer[i+1].second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
    i++;

    // all other levels
    for (; i < log2((float)SIZE)-1; i++) {
        hipExtent _size = images_size_pointer[i+1].first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        kernelConstructHPLevel<uint1, uint1><<<grid, block>>>(images_size_pointer[i].second , images_size_pointer[i+1].second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
}
