#include "hip/hip_runtime.h"
#include "gpu-mc.h"
#include "gpu-mc-kernel.h"
#include <utility>
#include <vector>

const unsigned int CUBESIZE = 8;
const unsigned int LOG2CUBESIZE = 3;
unsigned int SIZE;
unsigned int rawMemSize;
unsigned char * rawDataPtr;

// first level has char4 as datatype, which contains: (number of triangles, cube index, value of first cube element, 0)
// first to second level contain volumes with unsigned char as elements
// third to fifth (including) level contain unsigned short as elements
// sixth level and more uses int
std::vector<std::pair<unsigned int, void*> > images_size_pointer;

int isolevel = 50;

void setupCuda(unsigned char * voxels, unsigned int size) {
    SIZE = size;

    // Create images for the HistogramPyramid
    unsigned int bufferSize = SIZE;
    void * tmpDataPtr = NULL;
    // Make the two first buffers use INT8
    hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(uchar4));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    bufferSize /= 2;
    hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(unsigned char));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    bufferSize /= 2;
    // And the third, fourth and fifth INT16
    hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(unsigned short));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    bufferSize /= 2;
    hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(unsigned short));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    bufferSize /= 2;
    hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(unsigned short));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    bufferSize /= 2;
    // The rest will use INT32
    for(int i = 5; i < (log2((float)SIZE)); i++) {
        if(bufferSize == 1)
            bufferSize = 2; // Image cant be 1x1x1
        hipMalloc((void **) &tmpDataPtr, bufferSize * bufferSize * bufferSize * sizeof(unsigned int));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
        bufferSize /= 2;
    }

    // Transfer dataset to device
    rawMemSize = SIZE*SIZE*SIZE*sizeof(unsigned char);
    hipMalloc((void **) &rawDataPtr, rawMemSize);
    hipMemcpy(rawDataPtr, voxels, rawMemSize, hipMemcpyHostToDevice);
    delete[] voxels;
}

int log2(unsigned int val) {
    int log2Val = 0;
    while (val > 1) {
      val /= 2; log2Val++;
    }
    return log2Val;
}

void updateScalarField() {
    unsigned int _size = images_size_pointer[0].first;
    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((_size / CUBESIZE) * (_size / CUBESIZE), _size / CUBESIZE, 1);
    int log2GridSize = log2(_size / CUBESIZE);
    kernelClassifyCubes<<<grid , block>>>((uchar4 *)(images_size_pointer[0].second), rawDataPtr, isolevel, log2GridSize, _size/CUBESIZE-1, LOG2CUBESIZE, SIZE);
}
/*
void histoPyramidConstruction() {

        updateScalarField();

        // Run base to first level
		constructHPLevelKernel.setArg(0, images[0]);
		constructHPLevelKernel.setArg(1, images[1]);

        queue.enqueueNDRangeKernel(
			constructHPLevelKernel, 
			NullRange, 
			NDRange(SIZE/2, SIZE/2, SIZE/2), 
			NullRange
		);

        int previous = SIZE / 2;
        // Run level 2 to top level
        for(int i = 1; i < log2((float)SIZE)-1; i++) {
			constructHPLevelKernel.setArg(0, images[i]);
			constructHPLevelKernel.setArg(1, images[i+1]);
			previous /= 2;
            queue.enqueueNDRangeKernel(
				constructHPLevelKernel, 
				NullRange, 
				NDRange(previous, previous, previous), 
                NullRange
			);
        }
}
*/
