#include "hip/hip_runtime.h"
#include "gpu-mc.h"
#include "gpu-mc-kernel.h"
#include <utility>
#include <vector>
#include <iostream>
#include <cassert>
#include <cuda_gl_interop.h>

const unsigned int CUBESIZE = 8;
const unsigned int LOG2CUBESIZE = 3;
const unsigned int CUBESIZEHP = 2;
const unsigned int LOG2CUBESIZEHP = 1;
unsigned int SIZE;
unsigned int rawMemSize;
unsigned char * rawDataPtr;
struct hipGraphicsResource * vbo_cuda = NULL;
GLuint vbo_gl;
size_t vbo_size = 0;

unsigned int sum_of_triangles = 0;

// TODO How to use the VBO:
//      1. calc number of triangles
//      2. resize VBO to the correct size (triangles and normals)
//      3. calc the triangles
//      4. render

// first level has char4 as datatype, which contains: (number of triangles, cube index, value of first cube element, 0)
// first to second level contain volumes with unsigned char as elements
// third to fifth (including) level contain unsigned short as elements
// sixth level and more uses int
std::vector<std::pair<hipExtent, hipPitchedPtr> > images_size_pointer;

int isolevel = 50;

template<typename T>
T log2(T val) {
    T log2Val = 0;
    while (val > 1) {
      val /= 2; log2Val++;
    }
    return log2Val;
}

bool handleCudaError(const hipError_t& status) {
    std::string error_msg;
    switch (status) {
        case hipErrorInvalidValue: {
            error_msg = "hipErrorInvalidValue";
            break;
        }
        case hipErrorInvalidDevicePointer: {
            error_msg = "cudaErrorInvalidDevidePointer";
            break;
        }
        case hipErrorInvalidSymbol: {
            error_msg = "hipErrorInvalidSymbol";
            break;
        }
        case hipErrorInvalidMemcpyDirection: {
            error_msg = "hipErrorInvalidMemcpyDirection";
            break;
        }
        case hipErrorInvalidDevice: {
            error_msg = "hipErrorInvalidDevice";
            break;
        }
        case hipErrorSetOnActiveProcess: {
            error_msg = "hipErrorSetOnActiveProcess";
            break;
        }
        case hipErrorInvalidHandle: {
            error_msg = "hipErrorInvalidHandle";
            break;
        }
        case hipErrorOutOfMemory: {
            error_msg = hipErrorOutOfMemory;
            break;
        }
        case hipErrorUnknown: {
            error_msg = "hipErrorUnknown";
            break;
        }
        default: {
            error_msg = "unknown error";
            break;
        }
    }
    
    if (status != hipSuccess)
        std::cout << error_msg << std::endl;

    return status != hipSuccess;
}

void setupCuda(unsigned char * voxels, unsigned int size, GLuint vbo) {
    if (vbo == 0 && false) {
        std::cout << "Vertex Buffer Object to write into is invalid, I will exit" << std::endl;
        exit(1);
    }
    vbo_gl = vbo;
    handleCudaError(cudaGLSetGLDevice(0));

    SIZE = size;

    // Create images for the HistogramPyramid
    hipExtent bufferSize;
    hipPitchedPtr tmpDataPtr;
    // Make the two first buffers use INT8
    bufferSize.width = SIZE * sizeof(uchar4);
    bufferSize.height = SIZE;
    bufferSize.depth = SIZE;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    bufferSize.width = bufferSize.depth/2 * sizeof(uchar1);
    bufferSize.height = bufferSize.depth/2;
    bufferSize.depth = bufferSize.depth/2;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    // And the third, fourth and fifth INT16
    for (unsigned int i = 0; i < 3; i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(ushort1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        hipMalloc3D(&tmpDataPtr, bufferSize);
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // The rest will use INT32
    for(unsigned int i = 5; i < (log2(SIZE)); i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(uint1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        // Image cant be 1x1x1
        if (bufferSize.depth == 1) {
            bufferSize.width = 2 * sizeof(uint1);
            bufferSize.height = 2;
            bufferSize.depth = 2;
        }
        hipMalloc3D(&tmpDataPtr, bufferSize);
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // Transfer dataset to device
    rawMemSize = SIZE*SIZE*SIZE*sizeof(unsigned char);
    hipMalloc((void **) &rawDataPtr, rawMemSize);
    hipMemcpy(rawDataPtr, voxels, rawMemSize, hipMemcpyHostToDevice);
//    delete[] voxels;
}

// classifies each voxel and calculates the number of triangles needed for this
// voxel
void updateScalarField() {
    hipExtent _size = images_size_pointer.at(0).first;
    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((_size.depth / CUBESIZE) * (_size.depth / CUBESIZE), _size.depth / CUBESIZE, 1);
    int log2GridSize = log2(_size.depth / CUBESIZE);
    kernelClassifyCubes<<<grid , block>>>(images_size_pointer.at(0).second, rawDataPtr, isolevel, log2GridSize, _size.depth/CUBESIZE-1, LOG2CUBESIZE, _size.depth);
}

#ifdef DEBUG
// code to test classify cubes
unsigned int get_index(unsigned int x, unsigned int y, unsigned int z) {
  return x + y*SIZE + z*SIZE*SIZE;
}

void get_voxel_from_cube_id(unsigned int cube_id, unsigned int *x, unsigned int *y, unsigned *z) {
  // return lower left position of cube, other points can be obtained with +0,1
  *z = cube_id / (SIZE-1) / (SIZE-1);
  unsigned int cube_id_plane = cube_id % ((SIZE-1) * (SIZE-1));
  *y = cube_id_plane / (SIZE-1);
  *x = cube_id_plane % (SIZE-1);
}

int bit2Offset[] = {0, 1, 3, 2, 4, 5, 7, 6};
uint4 lokalCubeOffsets[8] = {
		{0, 0, 0, 0},
		{1, 0, 0, 0},
		{0, 0, 1, 0},
		{1, 0, 1, 0},
		{0, 1, 0, 0},
		{1, 1, 0, 0},
		{0, 1, 1, 0},
		{1, 1, 1, 0},
	}; 

unsigned char lokalNrOfTriangles[256] = {0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2, 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2, 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1, 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1, 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0};

template<typename T>
T* get_data_from_pitched_ptr(hipExtent size, hipPitchedPtr source) {
    T * lvl0_data = new T[size.depth*size.depth*size.depth];
    hipPitchedPtr h_pitched_ptr = make_hipPitchedPtr(lvl0_data, size.depth*sizeof(T), size.depth, size.depth);
    struct hipMemcpy3DParms parms = {0};
    parms.srcPtr = source;
    parms.dstPtr = h_pitched_ptr;
    parms.extent = size;
    parms.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&parms);
    return lvl0_data;
}

template<typename T>
T* get_data_from_pitched_ptr(std::pair<hipExtent, hipPitchedPtr> source) {
    return get_data_from_pitched_ptr<T>(source.first, source.second);
}

template<typename T>
T* get_data_from_pitched_ptr(unsigned int level) {
    return get_data_from_pitched_ptr<T>(images_size_pointer.at(level));
}

bool testUpdateScalarField(unsigned char * voxels) {
    updateScalarField();
    // get level0 data from gpu
    uchar4 * lvl0_data = get_data_from_pitched_ptr<uchar4>(0);

    sum_of_triangles = 0;

    // calc for each voxel index and number of triangles using a different implementation
    for (unsigned int i = 0; i < (SIZE-1)*(SIZE-1)*(SIZE-1); i++) {
        // get base voxel of the cube
        unsigned int x, y, z;
        get_voxel_from_cube_id(i, &x, &y, &z);  
        // look which vertices are below or above our threshold
        int lookuptable_index = 0;

        for (unsigned int id = 0; id < 8; id++) {
            uint4 offset = lokalCubeOffsets[bit2Offset[id]];
            unsigned char voxel = voxels[get_index(x + offset.x, y + offset.y, z + offset.z)];
            bool greater = voxel > isolevel;
            lookuptable_index |= greater << id;
        }
        unsigned int num_triangles = lokalNrOfTriangles[lookuptable_index];
        sum_of_triangles += num_triangles;

        // compare with results from gpu
        if (voxels[get_index(x, y, z)] != lvl0_data[get_index(x, y, z)].z) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "vertex value got from raw data is: " << static_cast<int>(voxels[get_index(x, y, z)]) << "\n value saved on gpu is: " << static_cast<int>(lvl0_data[get_index(x, y, z)].z) << std::endl;
            return false;
        }

        if (lookuptable_index != lvl0_data[get_index(x, y, z)].y) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "cube index calculated in software: " << lookuptable_index << "\ncube index calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].y)) << std::endl;
            return false;
        }

        if (num_triangles != lvl0_data[get_index(x, y, z)].x) {
            std::cout << "No match in number of triangles at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "number triangles calculated in software: " << num_triangles << "\nnumber of triangles calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].x)) << std::endl;
            return false;
        }
    }

    std::cout << "you will get " << sum_of_triangles << " triangles" << std::endl;

    delete [] lvl0_data;
    return true;
}
// end of code to test classifycubes
#endif // DEBUG

// calculates the total number of triangles needed
void histoPyramidConstruction() {
    dim3 block(CUBESIZEHP, CUBESIZEHP, CUBESIZEHP);
    
    // i=    0       1        2        3        4      5
    // uchar4, uchar1, ushort1, ushort1, ushort1, uint1, ...
    for (unsigned int i = 0; i < log2(SIZE)-1; i++) {
        hipExtent _size = images_size_pointer.at(i+1).first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        if (i == 0)
            // second level
            // uchar4 -> uchar1
            kernelConstructHPLevel<uchar4, uchar1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP);
        else if (i == 1)
            // third level
            // uchar1 -> ushort1
            kernelConstructHPLevel<uchar1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i == 2 || i == 3)
            // fourth, fifth level
            // ushort1 -> ushort1
            // ushort1 -> ushort1
            kernelConstructHPLevel<ushort1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i == 4)
            // sixth level
            // ushort1 -> uint1
            kernelConstructHPLevel<ushort1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else
            // all other levels
            // uint1 -> uint1
            kernelConstructHPLevel<uint1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
}

template<typename T>
unsigned int sum_3d_array(T const * const _3darray, const hipExtent& dim) {
    unsigned int sum = 0;
    for (unsigned int id = 0; id < dim.depth*dim.depth*dim.depth; id++) {
        sum+= _3darray[id].x;
    }
    return sum;
}

template<typename T>
unsigned int sum_3d_array(const std::pair<hipExtent, hipPitchedPtr>& pair) {
    T* sum_of_triangles_from_gpu = get_data_from_pitched_ptr<T>(pair);
    unsigned int sum = sum_3d_array(sum_of_triangles_from_gpu, pair.first);
    delete [] sum_of_triangles_from_gpu;
    return sum;
}

template<typename T>
unsigned int sum_3d_array(unsigned int level) {
    std::pair<hipExtent, hipPitchedPtr> pair = images_size_pointer.at(level);
    return sum_3d_array<T>(pair);
}

#ifdef DEBUG
template<typename T>
bool templatedTestHistoPyramidConstruction(unsigned int level) {
    unsigned int sum = sum_3d_array<T>(level);
    if (sum != sum_of_triangles) {
        std::cout << "at level " << level << std::endl;
        std::cout << "number of triangles calculated in software and hardware mismatches!" << std::endl;
        std::cout << "software: " << sum_of_triangles << ", hardware: " << sum << std::endl;
    }
    return sum == sum_of_triangles;
}

bool testHistoPyramidConstruction() {
    histoPyramidConstruction();
    bool success = true;
    for (unsigned int i = 0; i < log2(SIZE); i++) {
        if (i == 0)
            success &= templatedTestHistoPyramidConstruction<uchar4>(i);
        else if (i == 1)
            success &= templatedTestHistoPyramidConstruction<uchar1>(i);
        else if (i > 1 && i < 5)
            success &= templatedTestHistoPyramidConstruction<ushort1>(i);
        else
            success &= templatedTestHistoPyramidConstruction<uint1>(i);
    }
    return success;
}
#endif // DEBUG

void resizeVBO(size_t _vbo_size) {
    if (vbo_cuda != NULL) {
        handleCudaError(hipGraphicsUnregisterResource(vbo_cuda));
        vbo_cuda = NULL;
    }

    glBindBuffer(GL_ARRAY_BUFFER, vbo_gl);
    glBufferData(GL_ARRAY_BUFFER, _vbo_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    handleCudaError(hipGraphicsGLRegisterBuffer(&vbo_cuda, vbo_gl, cudaGraphicsMapFlagsWriteDiscard));
    
    vbo_size = _vbo_size;
}

// creates the VBO
int histoPyramidTraversal() {
    unsigned int sum = 0;
    assert(log2(SIZE) == images_size_pointer.size());
    size_t num_of_levels = images_size_pointer.size();
    std::pair<hipExtent, hipPitchedPtr> pair =  images_size_pointer.back();
    if (num_of_levels == 1)
        sum = sum_3d_array<uchar4>(pair);
    else if (num_of_levels == 2)
        sum = sum_3d_array<uchar1>(pair);
    else if (num_of_levels == 3 || num_of_levels == 4 || num_of_levels == 5)
        sum = sum_3d_array<ushort1>(pair);
    else
        sum = sum_3d_array<uint1>(pair);
    sum_of_triangles = sum;
    std::cout << "you will get " << sum << " triangles" << std::endl;

    handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("num_of_levels"), &num_of_levels, sizeof(size_t), 0, hipMemcpyHostToDevice));

    for (unsigned int i = 0; i < num_of_levels; i++) {
        handleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("levels"), &(images_size_pointer.at(i).second), sizeof(hipPitchedPtr), i*sizeof(hipPitchedPtr), hipMemcpyHostToDevice));
    }
    // TODO to get this working I need to setup OpenGL with VBO
    //      since OpenGL over SSH is hard, maybe I can just write into an array
    // resize buffer
    // normals, triangles, three coordinates, three points in float
    size_t buffer_size = sum*2*3*3*sizeof(float);
    // just increasing the buffer would be enough as well, but atm this is easier
    if (buffer_size > vbo_size)
        resizeVBO(buffer_size);

    float3 * triangle_data = NULL;
    hipGraphicsMapResources(1, &vbo_cuda, 0);
    size_t num_bytes = 0;
    hipGraphicsResourceGetMappedPointer((void**)&triangle_data, &num_bytes, vbo_cuda);
    assert(num_bytes >= buffer_size);

    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((pair.first.depth / CUBESIZE) * (pair.first.depth / CUBESIZE), pair.first.depth / CUBESIZE, 1);
    int log2GridSize = log2(pair.first.depth / CUBESIZE);
    
    traverseHP<<<grid, block>>>(
        triangle_data,
        isolevel,
        sum,
        log2GridSize, pair.first.depth/CUBESIZE-1, LOG2CUBESIZE, 
        pair.first.depth
        );
    
    hipGraphicsUnmapResources(1, &vbo_cuda, 0);
    return sum;
}

#ifdef DEBUG
bool operator==(const hipPitchedPtr& cpp1, const hipPitchedPtr& cpp2) {
    return cpp1.pitch == cpp2.pitch && cpp1.ptr == cpp2.ptr && cpp1.xsize == cpp2.xsize && cpp1.ysize == cpp2.ysize;
}

bool testCudaPitchedPtrOnDevice() {
    std::pair<hipExtent, hipPitchedPtr> pair = images_size_pointer.back();
    dim3 grid(1,1,1);
    dim3 block(1,1,1);
    unsigned int h_count = 0;
    unsigned int * d_count;
    hipMalloc(&d_count, sizeof(unsigned int));
    sum_values_of_pitched_ptr<<<grid, block>>>(d_count, pair.first.depth, log2(pair.first.depth));
    hipMemcpy(&h_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(d_count);
    bool success = sum_of_triangles == h_count;
    if (!success)
        std::cout << "something is wrong with the hipPitchedPtr copied to the GPU via hipMemcpyToSymbol" << std::endl;
    return success;
}

bool testHistoPyramidTraversal() {
    histoPyramidTraversal();
    bool success = true;
    size_t num_of_levels_readback = 0;
    handleCudaError(hipMemcpyFromSymbol(&num_of_levels_readback, HIP_SYMBOL("num_of_levels"), sizeof(size_t), 0, hipMemcpyDeviceToHost));

    success &= images_size_pointer.size() == num_of_levels_readback;
    if (!success) {
        std::cout << "number of levels on GPU are not set correctly: " << num_of_levels_readback << ", should be: " << images_size_pointer.size() << std::endl;
    }

    hipPitchedPtr cpp = {0};
    for (unsigned int i = 0; i < num_of_levels_readback; i++) {
        handleCudaError(hipMemcpyFromSymbol(&cpp, HIP_SYMBOL("levels"), sizeof(hipPitchedPtr), i*sizeof(hipPitchedPtr), hipMemcpyDeviceToHost));
        bool tmp_success = cpp == images_size_pointer.at(i).second;
        if (!tmp_success) {
            std::cout << "hipPitchedPtr into arguments of GPU does not match at level " << i << std::endl;
        }
        success &= tmp_success;
    }
    success &= testCudaPitchedPtrOnDevice();
    return success;
}


bool runTests(unsigned char * voxels) {
  bool success = testUpdateScalarField(voxels);
  success &= testHistoPyramidConstruction();
  success &= testHistoPyramidTraversal();
  return success;
}
#endif // DEBUG

int marching_cube(int _isolevel) {
    if (isolevel != _isolevel) {
        isolevel = _isolevel; 
        // first level
        updateScalarField();
        // all other levels
        histoPyramidConstruction();
        histoPyramidTraversal();
    }
    return sum_of_triangles;
}
