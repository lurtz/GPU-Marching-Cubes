#include "hip/hip_runtime.h"
#include "gpu-mc.h"
#include "gpu-mc-kernel.h"
#include <utility>
#include <vector>
#include <iostream>

const unsigned int CUBESIZE = 8;
const unsigned int LOG2CUBESIZE = 3;
const unsigned int CUBESIZEHP = 2;
const unsigned int LOG2CUBESIZEHP = 1;
unsigned int SIZE;
unsigned int rawMemSize;
unsigned char * rawDataPtr;

#ifdef DEBUG
unsigned int sum_of_triangles = 0;
#endif

// first level has char4 as datatype, which contains: (number of triangles, cube index, value of first cube element, 0)
// first to second level contain volumes with unsigned char as elements
// third to fifth (including) level contain unsigned short as elements
// sixth level and more uses int
std::vector<std::pair<hipExtent, hipPitchedPtr> > images_size_pointer;

int isolevel = 50;

template<typename T>
T log2(T val) {
    T log2Val = 0;
    while (val > 1) {
      val /= 2; log2Val++;
    }
    return log2Val;
}

bool handleCudaError(const hipError_t& status) {
    if (status == hipErrorInvalidValue) {
        std::cout << "hipErrorInvalidValue" << std::endl;
    }
    if (status == hipErrorInvalidDevicePointer) {
        std::cout << "cudaErrorInvalidDevidePointer" << std::endl;
    }
    return status != hipSuccess;
}

void setupCuda(unsigned char * voxels, unsigned int size) {
    SIZE = size;

    // Create images for the HistogramPyramid
    hipExtent bufferSize;
    hipPitchedPtr tmpDataPtr;
    // Make the two first buffers use INT8
    bufferSize.width = SIZE * sizeof(uchar4);
    bufferSize.height = SIZE;
    bufferSize.depth = SIZE;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    bufferSize.width = bufferSize.depth/2 * sizeof(uchar1);
    bufferSize.height = bufferSize.depth/2;
    bufferSize.depth = bufferSize.depth/2;
    hipMalloc3D(&tmpDataPtr, bufferSize);
    handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
    images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));

    // And the third, fourth and fifth INT16
    for (unsigned int i = 0; i < 3; i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(ushort1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        hipMalloc3D(&tmpDataPtr, bufferSize);
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // The rest will use INT32
    for(unsigned int i = 5; i < (log2(SIZE)); i++) {
        bufferSize.width = bufferSize.depth/2 * sizeof(uint1);
        bufferSize.height = bufferSize.depth/2;
        bufferSize.depth = bufferSize.depth/2;
        // Image cant be 1x1x1
        if (bufferSize.depth == 1) {
            bufferSize.width = 2 * sizeof(uint1);
            bufferSize.height = 2;
            bufferSize.depth = 2;
        }
        hipMalloc3D(&tmpDataPtr, bufferSize);
        handleCudaError(hipMemset3D(tmpDataPtr, 0, bufferSize));
        images_size_pointer.push_back(std::make_pair(bufferSize, tmpDataPtr));
    }

    // Transfer dataset to device
    rawMemSize = SIZE*SIZE*SIZE*sizeof(unsigned char);
    hipMalloc((void **) &rawDataPtr, rawMemSize);
    hipMemcpy(rawDataPtr, voxels, rawMemSize, hipMemcpyHostToDevice);
//    delete[] voxels;
}

void updateScalarField() {
    hipExtent _size = images_size_pointer.at(0).first;
    dim3 block(CUBESIZE, CUBESIZE, CUBESIZE);
    dim3 grid((_size.depth / CUBESIZE) * (_size.depth / CUBESIZE), _size.depth / CUBESIZE, 1);
    int log2GridSize = log2(_size.depth / CUBESIZE);
    kernelClassifyCubes<<<grid , block>>>(images_size_pointer.at(0).second, rawDataPtr, isolevel, log2GridSize, _size.depth/CUBESIZE-1, LOG2CUBESIZE, _size.depth);
}

#ifdef DEBUG
// code to test classify cubes
unsigned int get_index(unsigned int x, unsigned int y, unsigned int z) {
  return x + y*SIZE + z*SIZE*SIZE;
}

void get_voxel_from_cube_id(unsigned int cube_id, unsigned int *x, unsigned int *y, unsigned *z) {
  // return lower left position of cube, other points can be obtained with +0,1
  *z = cube_id / (SIZE-1) / (SIZE-1);
  unsigned int cube_id_plane = cube_id % ((SIZE-1) * (SIZE-1));
  *y = cube_id_plane / (SIZE-1);
  *x = cube_id_plane % (SIZE-1);
}

int bit2Offset[] = {0, 1, 3, 2, 4, 5, 7, 6};
uint4 lokalCubeOffsets[8] = {
		{0, 0, 0, 0},
		{1, 0, 0, 0},
		{0, 0, 1, 0},
		{1, 0, 1, 0},
		{0, 1, 0, 0},
		{1, 1, 0, 0},
		{0, 1, 1, 0},
		{1, 1, 1, 0},
	}; 

unsigned char lokalNrOfTriangles[256] = {0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2, 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2, 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1, 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1, 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0};

template<typename T>
T* get_data_from_pitched_ptr(hipExtent size, hipPitchedPtr source) {
    T * lvl0_data = new T[size.depth*size.depth*size.depth];
    hipPitchedPtr h_pitched_ptr = make_hipPitchedPtr(lvl0_data, size.depth*sizeof(T), size.depth, size.depth);
    struct hipMemcpy3DParms parms = {0};
    parms.srcPtr = source;
    parms.dstPtr = h_pitched_ptr;
    parms.extent = size;
    parms.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&parms);
    return lvl0_data;
}

template<typename T>
T* get_data_from_pitched_ptr(std::pair<hipExtent, hipPitchedPtr> source) {
    return get_data_from_pitched_ptr<T>(source.first, source.second);
}

template<typename T>
T* get_data_from_pitched_ptr(unsigned int level) {
    return get_data_from_pitched_ptr<T>(images_size_pointer.at(level));
}

bool testUpdateScalarField(unsigned char * voxels) {
    // get level0 data from gpu
    uchar4 * lvl0_data = get_data_from_pitched_ptr<uchar4>(0);

    sum_of_triangles = 0;

    // calc for each voxel index and number of triangles using a different implementation
    for (unsigned int i = 0; i < (SIZE-1)*(SIZE-1)*(SIZE-1); i++) {
        // get base voxel of the cube
        unsigned int x, y, z;
        get_voxel_from_cube_id(i, &x, &y, &z);  
        // look which vertices are below or above our threshold
        int lookuptable_index = 0;

        for (unsigned int id = 0; id < 8; id++) {
            uint4 offset = lokalCubeOffsets[bit2Offset[id]];
            unsigned char voxel = voxels[get_index(x + offset.x, y + offset.y, z + offset.z)];
            bool greater = voxel > isolevel;
            lookuptable_index |= greater << id;
        }
        unsigned int num_triangles = lokalNrOfTriangles[lookuptable_index];
        sum_of_triangles += num_triangles;

        // compare with results from gpu
        if (voxels[get_index(x, y, z)] != lvl0_data[get_index(x, y, z)].z) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "vertex value got from raw data is: " << static_cast<int>(voxels[get_index(x, y, z)]) << "\n value saved on gpu is: " << static_cast<int>(lvl0_data[get_index(x, y, z)].z) << std::endl;
            return false;
        }

        if (lookuptable_index != lvl0_data[get_index(x, y, z)].y) {
            std::cout << "No match at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "cube index calculated in software: " << lookuptable_index << "\ncube index calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].y)) << std::endl;
            return false;
        }

        if (num_triangles != lvl0_data[get_index(x, y, z)].x) {
            std::cout << "No match in number of triangles at position: (" << x << ", " << y << ", " << z << ")" << std::endl;
            std::cout << "number triangles calculated in software: " << num_triangles << "\nnumber of triangles calculated in hardware: " << static_cast<int>((lvl0_data[get_index(x, y, z)].x)) << std::endl;
            return false;
        }
    }

    std::cout << "you will get " << sum_of_triangles << " triangles" << std::endl;

    delete [] lvl0_data;
    return true;
}
// end of code to test classifycubes
#endif // DEBUG

void histoPyramidConstruction() {
    // first level
    updateScalarField();

    dim3 block(CUBESIZEHP, CUBESIZEHP, CUBESIZEHP);
    
    for (unsigned int i = 0; i < log2(SIZE)-1; i++) {
        hipExtent _size = images_size_pointer.at(i+1).first;
        dim3 grid((_size.depth / CUBESIZEHP) * (_size.depth / CUBESIZEHP), _size.depth / CUBESIZEHP, 1);
        int log2GridSize = log2(_size.depth / CUBESIZEHP);
        if (i == 0)
            // second level
            kernelConstructHPLevel<uchar4, uchar1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP);
        else if (i == 1)
            // third level
            kernelConstructHPLevel<uchar1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i > 1 && i < 4)
            // fourth, fifth level
            kernelConstructHPLevel<ushort1, ushort1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else if (i == 5)
            // sixth level
            kernelConstructHPLevel<ushort1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
        else
            // all other levels
            kernelConstructHPLevel<uint1, uint1><<<grid, block>>>(images_size_pointer.at(i).second , images_size_pointer.at(i+1).second, log2GridSize, _size.depth/CUBESIZEHP-1, LOG2CUBESIZEHP); 
    }
}

#ifdef DEBUG
template<typename T>
bool templatedTestHistoPyramidConstruction(unsigned int level) {
    std::pair<hipExtent, hipPitchedPtr> pair = images_size_pointer.at(level);
    T* sum_of_triangles_from_gpu = get_data_from_pitched_ptr<T>(pair);
    unsigned int sum = 0;
    for (unsigned int id = 0; id < pair.first.depth*pair.first.depth*pair.first.depth; id++) {
        sum+= sum_of_triangles_from_gpu[id].x;
    }
    delete [] sum_of_triangles_from_gpu;
    sum_of_triangles_from_gpu = 0;
    if (sum != sum_of_triangles) {
        std::cout << "at level " << level << std::endl;
        std::cout << "number of triangles calculated in software and hardware mismatches!" << std::endl;
        std::cout << "software: " << sum_of_triangles << ", hardware: " << sum << std::endl;
    }
    return sum == sum_of_triangles;
}

bool testHistoPyramidConstruction() {
    bool success = true;
    for (unsigned int i = 0; i < log2(SIZE); i++) {
        if (i == 0)
            success &= templatedTestHistoPyramidConstruction<uchar4>(i);
        else if (i == 1)
            success &= templatedTestHistoPyramidConstruction<uchar1>(i);
        else if (i > 1 && i < 5)
            success &= templatedTestHistoPyramidConstruction<ushort1>(i);
        else
            success &= templatedTestHistoPyramidConstruction<uint1>(i);
    }
    return success;
}
#endif // DEBUG
